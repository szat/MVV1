#include "hip/hip_runtime.h"
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>

#include <string>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <conio.h>
#include <ctime>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>
#include "binary_read.h"

using namespace std;
using namespace cv;

typedef std::chrono::high_resolution_clock Clock;

__global__
void kernel2D(uchar* d_output, uchar* d_input, int w, int h, float * d_affineData)
{
	int c = blockIdx.x*blockDim.x + threadIdx.x;
	int r = blockIdx.y*blockDim.y + threadIdx.y;
	int i = r * w + c;

	if ((r >= h) || (c >= w)) return;

	int new_c = (int)(d_affineData[0] * (float)c + d_affineData[1] * (float)r + d_affineData[2]);
	int new_r = (int)(d_affineData[3] * (float)c + d_affineData[4] * (float)r + d_affineData[5]);

	if ((new_r >= h) || (new_c >= w) || (new_r < 0) || (new_c < 0)) return;

	int new_i = new_r * w + new_c;
	d_output[new_i] = d_input[i];
}

__global__
void kernel2D_subpix(uchar4* d_output, uchar4* d_input, short* d_raster1, int w, int h, float * d_affineData, int subDiv, float tau, bool reverse)
{
	if (tau > 1 || tau < 0) return;

	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int raster_index = (row * w + col);
	//int color_index = raster_index * 3;

	// should not need to do this check if everything is good, must be an extra pixel
	if (raster_index >= w * h) return;
	if ((row >= h) || (col >= w)) return;

	short affine_index = d_raster1[raster_index];
	short offset = (affine_index - 1) * 12;
	if (reverse) {
		offset += 6;
	}
	if (affine_index != 0) {
		// triangle indexes start at 1
		float diff = 1 / (float)subDiv;
		for (int i = 0; i < subDiv; i++) {
			for (int j = 0; j < subDiv; j++) {
				int new_c = (int)(((1 - tau) + tau*d_affineData[offset]) * (float)(col - 0.5 + (diff * i)) + (tau * d_affineData[offset + 1]) * (float)(row - 0.5 + (diff * j)) + (tau * d_affineData[offset + 2]));
				int new_r = (int)((tau * d_affineData[offset + 3]) * (float)(col - 0.5 + (diff * i)) + ((1 - tau) + tau * d_affineData[offset + 4]) * (float)(row - 0.5 + (diff * j)) + (tau * d_affineData[offset + 5]));
				if ((new_r >= h) || (new_c >= w) || (new_r < 0) || (new_c < 0)) return;
				int new_i = new_r * w + new_c;
				d_output[new_i] = d_input[raster_index];
			}
		}
	}


}

__global__
void kernel2D_add(uchar4* d_output, uchar4* d_input_1, uchar4* d_input_2, int w, int h, float tau) {
	//tau is from a to b
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int raster_index = (row * w + col);

	// should not need to do this check if everything is good, must be an extra pixel
	if (raster_index >= w * h) return;
	if ((row >= h) || (col >= w)) return;


	if (d_input_1[raster_index].x == 0 && d_input_1[raster_index].y == 0 && d_input_1[raster_index].z == 0) {
		d_output[raster_index] = d_input_2[raster_index];
	}
	else if (d_input_2[raster_index].x == 0 && d_input_2[raster_index].y == 0 && d_input_2[raster_index].z == 0) {
		d_output[raster_index] = d_input_1[raster_index];
	}
	else {
		d_output[raster_index].x = tau*d_input_1[raster_index].x + (1 - tau)*d_input_2[raster_index].x;
		d_output[raster_index].y = tau*d_input_1[raster_index].y + (1 - tau)*d_input_2[raster_index].y;
		d_output[raster_index].z = tau*d_input_1[raster_index].z + (1 - tau)*d_input_2[raster_index].z;
	}
}

Mat trial_binary_render(uchar4 *image, int width, int height) {
	Mat img(height, width, CV_8UC3, Scalar(0, 0, 0));
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {

			int index = i * width + j;

			uchar r = image[index].x;
			uchar g = image[index].y;
			uchar b = image[index].z;

			Vec3b color = Vec3b(r, g, b);

			img.at<Vec3b>(i, j) = color;
		}
	}

	return img;
}

void uchar4_test() {
	uchar *test_uchar = new uchar[4];
	test_uchar[0] = (uchar)25;
	test_uchar[1] = (uchar)50;
	test_uchar[2] = (uchar)100;
	test_uchar[3] = (uchar)0;
	test_uchar[4] = (uchar)10;
	test_uchar[5] = (uchar)20;
	test_uchar[6] = (uchar)30;
	test_uchar[7] = (uchar)0;

	uchar4 *test_uchar4 = new uchar4[0];
	memcpy(test_uchar4, test_uchar, 8);

	uchar4 test1 = test_uchar4[0];
	uchar4 test2 = test_uchar4[1];

	int size = sizeof(uchar4);


	cout << "test";
}

int main(int argc, char ** argv) {
	uchar4_test();

	cout << "welcome to cuda_demo testing unit!" << endl;
	cout << "loading 2 images with openCV, processing and adding them with cuda (grayscale)." << endl;

	// Initializing CUDA
	uchar *h_tester = new uchar[1];
	h_tester[0] = (uchar)0;
	uchar *d_tester;
	hipMalloc((void**)&d_tester, sizeof(uchar));
	hipMemcpy(d_tester, h_tester, sizeof(uchar), hipMemcpyHostToDevice);
	hipFree(d_tester);

	auto t1 = std::chrono::high_resolution_clock::now();

	string img_path_1 = "../../data_store/binary/david_1.bin";
	string img_path_2 = "../../data_store/binary/david_2.bin";
	string raster1_path = "../../data_store/raster/rasterA.bin";
	string raster2_path = "../../data_store/raster/rasterB.bin";
	string affine_path = "../../data_store/affine/affine_1.bin";

	// BINARY IMAGE READ
	int length_1 = 0;
	int length_2 = 0;
	int width_1 = 0;
	int width_2 = 0;
	int height_1 = 0;
	int height_2 = 0;
	uchar4 *h_in_1 = read_uchar4_array(img_path_1, length_1, width_1, height_1);
	uchar4 *h_in_2 = read_uchar4_array(img_path_2, length_2, width_2, height_2);


	// RASTER READ
	int num_pixels_1 = 0;
	int num_pixels_2 = 0;
	short *h_raster1 = read_short_array(raster1_path, num_pixels_1);
	short *h_raster2 = read_short_array(raster2_path, num_pixels_2);

	// AFFINE READ
	int num_floats = 0;
	float *h_affine_data = read_float_array(affine_path, num_floats);
	int num_triangles = num_floats / 12;

	if (height_1 != height_2 || width_1 != width_2) {
		cout << "Incompatible image sizes. Program will now crash.\n";
		exit(-1);
	}

	int W = width_1;
	int H = height_1;
	int mem_alloc = W * H * 4 * sizeof(uchar);

	uchar4 *h_out_1;
	uchar4 *h_out_2;
	uchar4 *h_sum;

	// there must be a faster way to initialize these arrays to all zeros
	uchar *zeros = new uchar[mem_alloc];
	for (int j = 0; j < mem_alloc; j++) zeros[j] = 0;
	h_out_1 = (uchar4*)malloc(mem_alloc);
	h_out_2 = (uchar4*)malloc(mem_alloc);
	h_sum = (uchar4*)malloc(mem_alloc);
	memcpy(h_out_1, zeros, mem_alloc);
	memcpy(h_out_2, zeros, mem_alloc);
	memcpy(h_sum, zeros, mem_alloc);

	//--Sending the data to the GPU memory
	cout << "declaring device data-structures..." << endl;

	float * d_affine_data;
	hipMalloc((void**)&d_affine_data, num_floats * sizeof(float));
	hipMemcpy(d_affine_data, h_affine_data, num_floats * sizeof(float), hipMemcpyHostToDevice);

	short *d_raster1;
	hipMalloc((void**)&d_raster1, W * H * sizeof(short));
	hipMemcpy(d_raster1, h_raster1, W * H * sizeof(short), hipMemcpyHostToDevice);

	short *d_raster2;
	hipMalloc((void**)&d_raster2, W * H * sizeof(short));
	hipMemcpy(d_raster2, h_raster2, W * H * sizeof(short), hipMemcpyHostToDevice);

	uchar4 * d_in_1;
	hipMalloc((void**)&d_in_1, mem_alloc);
	hipMemcpy(d_in_1, h_in_1, mem_alloc, hipMemcpyHostToDevice);

	uchar4 * d_out_1;
	hipMalloc((void**)&d_out_1, mem_alloc);
	hipMemcpy(d_out_1, h_out_1, mem_alloc, hipMemcpyHostToDevice);

	uchar4 * d_in_2;
	hipMalloc((void**)&d_in_2, mem_alloc);
	hipMemcpy(d_in_2, h_in_2, mem_alloc, hipMemcpyHostToDevice);

	uchar4 * d_out_2;
	hipMalloc((void**)&d_out_2, mem_alloc);
	hipMemcpy(d_out_2, h_out_2, mem_alloc, hipMemcpyHostToDevice);

	uchar4 * d_sum;
	hipMalloc((void**)&d_sum, mem_alloc);
	hipMemcpy(d_sum, h_sum, mem_alloc, hipMemcpyHostToDevice);

	dim3 blockSize(32, 32);
	int bx = (W + 32 - 1) / 32;
	int by = (H + 32 - 1) / 32;
	dim3 gridSize = dim3(bx, by);

	float tau = 0.5f;
	float reverse_tau = 1.0f - tau;
	int reversal_offset = 0;


	kernel2D_subpix << <gridSize, blockSize >> >(d_out_1, d_in_1, d_raster1, W, H, d_affine_data, 4, tau, false);
	kernel2D_subpix << <gridSize, blockSize >> >(d_out_2, d_in_2, d_raster2, W, H, d_affine_data, 4, reverse_tau, true);
	kernel2D_add << <gridSize, blockSize >> > (d_sum, d_out_1, d_out_2, W, H, tau);


	hipMemcpy(h_out_1, d_out_1, mem_alloc, hipMemcpyDeviceToHost);
	hipMemcpy(h_out_2, d_out_2, mem_alloc, hipMemcpyDeviceToHost);
	hipMemcpy(h_sum, d_sum, mem_alloc, hipMemcpyDeviceToHost);

	hipFree(d_in_1);
	hipFree(d_out_1);
	hipFree(d_raster1);
	hipFree(d_in_2);
	hipFree(d_out_2);
	hipFree(d_raster2);
	hipFree(d_affine_data);
	hipFree(d_sum);

	auto t2 = std::chrono::high_resolution_clock::now();
	std::cout << "write short took "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count()
		<< " milliseconds\n";

	//trial_binary_render(h_sum, W, H);
	Mat img1_initial = trial_binary_render(h_in_1, W, H);
	Mat img2_initial = trial_binary_render(h_in_2, W, H);
	Mat img1_final = trial_binary_render(h_out_1, W, H);
	Mat img2_final = trial_binary_render(h_out_2, W, H);
	Mat img_final = trial_binary_render(h_sum, W, H);

	return 0;
}