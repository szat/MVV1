#include "hip/hip_runtime.h"
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>

#include <string>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <conio.h>
#include <ctime>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>

using namespace std;
using namespace cv;

typedef std::chrono::high_resolution_clock Clock;

//load 2d array
//do something on 2d array
//send back the 2d array

__global__ void cube(float * d_out, float * d_in) {
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f*f*f;
}

__device__ int flatten_idx_gpu(int * matrixIdx, int numRows, int numCols) {
	if (0 <= matrixIdx[0] < numRows && 0 <= matrixIdx[1] < numCols) return matrixIdx[0] * numCols + matrixIdx[1];
	else return 0;
}

__device__ int * unflatten_idx_gpu(int arrayIdx, int numRows, int numCols) {
	if (0 <= arrayIdx < numRows*numCols) {
		int rowIdx = arrayIdx / numCols;
		int idx[] = { rowIdx ,  arrayIdx - rowIdx*numCols };
		return idx;
	}
	return 0;
}

__global__ void rowOperation_block(float * d_matrix1D_out, float * d_matrix1D_in, int numRows, int numCols) {
	//number of blocks should be the number of rows
	//number of threads should be the number of cols
	int idxBlk = blockIdx.x;
	int idxThd = threadIdx.x;
	//give a row per block, in a 2d matrix it would be mat[idxBlk][idxThd], mat[idxBlk][idxThd], mat[idxBlk][idxThd], ...
	int idx2D[] = { idxBlk, idxThd };
	int matIdx = flatten_idx_gpu(idx2D, numRows, numCols);
	d_matrix1D_out[matIdx] = d_matrix1D_in[matIdx] + 1;
}

__global__ void colOperation_block(float * d_matrix1D_out, float * d_matrix1D_in, int numRows, int numCols) {
	int idxBlk = blockIdx.x;
	int idxThd = threadIdx.x;
	//give a column per thread, in a 2d matrix it would be mat[idxThd][idxBlk], mat[idxThd][idxBlk], mat[idxThd][idxBlk], ...
	int idx2D[] = { idxThd, idxBlk };
	int matIdx = flatten_idx_gpu(idx2D, numRows, numCols);
	d_matrix1D_out[matIdx] = d_matrix1D_in[matIdx] + 0.01;
}

__global__ void rowOperation(float * d_matrix1D_out, float * d_matrix1D_in, int numRows, int numCols) {
	int idx = threadIdx.x;
	//printf("We are in thread %d\n", idx);
	//give a row per thread, in a 2d matrix it would be mat[idx][0], mat[idx][1], mat[idx][2], ...
	for (int col = 0; col < numCols; col++) {
		int idx2D[] = { idx, col };
		int matIdx = flatten_idx_gpu(idx2D, numRows, numCols);
		int matIdx_out = 3 * (matIdx + 32) % numCols;
		d_matrix1D_out[matIdx_out] = d_matrix1D_in[matIdx] + 1; //in our sample code should increment the rows from 2.14 to 3.14 for instance
	}
}

__global__ void image_to_inter_1C(unsigned char * d_matrix1D_out, unsigned char * d_matrix1D_in, int numRows, int numCols, float tau) {
	int rowIdx = blockIdx.x; //give a row per block ==> numRows
	int colIdx = threadIdx.x; //give a col per thread, thus a pixel per thread ==> numCols
	//if (rowIdx < numRows && colIdx < numCols && 0 <= tau && tau <= 1) {
		int idx2D_in[] = { rowIdx, colIdx };
		int matIdx_in = flatten_idx_gpu(idx2D_in, numRows, numCols);
		d_matrix1D_out[matIdx_in] = d_matrix1D_in[matIdx_in];
		//d_matrix1D_out[matIdx_in] = (unsigned char) (tau * (short) d_matrix1D_in[matIdx_in]);
		/*
		int matIdx_in = flatten_idx_gpu(idx2D_in, numRows, numCols);
		int triangleIdx = d_matrixTriangles1D_in[matIdx_in];
		if (triangleIdx != -1) {
			float triangleData[6];
			for (int i = 0; i < 6; i++) triangleData[i] = d_triangleData1D_in[triangleIdx + i];
			//using triangleData project pixel in d_matrix1D_in at position matIdx_in onto pixel in d_matrix1D_out at position matIdx_out
			int idx2D_out[] = { (int)rowIdx*tau, (int)colIdx*tau };
			int matIdx_out = flatten_idx_gpu(idx2D_out, numRows, numCols);
			d_matrix1D_out[matIdx_out] = d_matrix1D_in[matIdx_in];
		}
		*/
	//}
}

__global__ void colOperation(float * d_matrix1D_out, float * d_matrix1D_in, int numRows, int numCols) {
	int idx = threadIdx.x;
	//give a column per thread, in a 2d matrix it would be mat[0][idx], mat[1][idx], mat[2][idx], ...
	for (int row = 0; row < numRows; row++) {
		int idx2D[] = { row, idx };
		int matIdx = flatten_idx_gpu(idx2D, numRows, numCols);
		d_matrix1D_out[matIdx] = d_matrix1D_in[matIdx] + 0.01; //in ou sample code should increment the cols from 2.14 to 2.15
	}
}

int flatten_idx(int * matrixIdx, int numRows, int numCols) {
	if (0 <= matrixIdx[0] < numRows && 0 <= matrixIdx[1] < numCols) return matrixIdx[0] * numCols + matrixIdx[1];
	else return 0;
}

int * unflatten_idx(int arrayIdx, int numRows, int numCols) {
	if (0 <= arrayIdx < numRows*numCols) {
		int rowIdx = arrayIdx / numCols;
		int idx[] = { rowIdx ,  arrayIdx - rowIdx*numCols };
		return idx;
	}
	return 0;
}

__global__ void compute2D(uchar4 * imageData_out, uchar4 * imageData_in, float tau) {
	int c = blockIdx.x*blockDim.x + threadIdx.x;
	int r = blockIdx.y*blockDim.y + threadIdx.y;

}

__global__
void kernel2D(uchar *d_output, uchar* d_input, int w, int h, float tau)
{
	int c = blockIdx.x*blockDim.x + threadIdx.x;
	int r = blockIdx.y*blockDim.y + threadIdx.y;
	int i = r * w + c; 
	
	if ((r >= h) || (c >= w)) return;

	d_output[i] = d_input[i] / 2;
	/*
	d_output[i].x = (uchar) tau * d_input[i].x;    //Compute red
	d_output[i].y = (uchar) tau * d_input[i].y; //Compute green
	d_output[i].z = (uchar) tau * d_input[i].z;  //Compute blue
	d_output[i].w = 255; // Fully 
	*/
}

int main(int argc, char ** argv) {
	cout << "Welcome to cuda_demo testing unit!" << endl;
	cout << "Loading one image with openCV! (grayscale)" << endl;

	string address1 = "..\\data_store\\big_picture.jpg";
	Mat img1 = imread(address1, IMREAD_GRAYSCALE);
	const int W = img1.size().width;
	const int H = img1.size().height;
	const int ARRAY_BYTES = W*H * sizeof(uchar);

	uchar *h_in;
	uchar *h_out;
	h_in = (uchar*)malloc(ARRAY_BYTES);
	h_out = (uchar*)malloc(ARRAY_BYTES);

	Mat img1Flat = img1.reshape(1, 1);
	h_in = img1Flat.data;

	uchar * d_in;
	uchar * d_out;
	hipMalloc((void**)&d_in, ARRAY_BYTES);
	hipMalloc((void**)&d_out, ARRAY_BYTES);

	const dim3 blockSize(32, 32);

	const int bx = (W + 32 - 1) / 32;
	const int by = (H + 32 - 1) / 32;
	const dim3 gridSize = dim3(bx, by);

	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	kernel2D << <gridSize, blockSize >> >(d_out, d_in, W, H, 0.3);	
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);

	Mat out = Mat(1, W*H, CV_8UC1, h_out);
	out = out.reshape(1, H);


	return 0;
	/*


	const int W = img1.size().width;
	const int H = img1.size().height;

	dim3 blockSize(32, 32); 
	
	int bx = (W + blockSize.x - 1) / blockSize.x;
	int by = (H + blockSize.y - 1) / blockSize.y;
	dim3 gridSize = dim3(bx, by);

	const int numRows = img1.rows;
	const int numCols = img1.cols;

	const int ARRAY_SIZE = numRows * numCols;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);

	unsigned char *h_img1Data_in;
	unsigned char *h_img1Data_out;
	h_img1Data_out = (unsigned char *)malloc(ARRAY_BYTES);
	h_img1Data_in = (unsigned char *)malloc(ARRAY_BYTES);
	
	Mat img1Flat = img1.reshape(1, 1);
	h_img1Data_in = img1Flat.data;

	//for(int i = 0; i < ARRAY_SIZE; i++) h_img1Data_out[i] = 0; //just to put something in it

	unsigned char * d_img1Data_in;
	unsigned char * d_img1Data_out;

	hipMalloc((void**)&d_img1Data_in, ARRAY_BYTES);
	hipMalloc((void**)&d_img1Data_out, ARRAY_BYTES);

	cout << "number of bytes " << ARRAY_BYTES << endl;

	for (int i = 0; i < 1; i++) {
		hipMemcpy(d_img1Data_in, h_img1Data_in, ARRAY_BYTES, hipMemcpyHostToDevice);
		auto t1 = Clock::now();
		image_to_inter_1C << <numRows, numCols >> > (d_img1Data_out, d_img1Data_in, numRows, numCols, 0.7);
		hipMemcpy(h_img1Data_out, d_img1Data_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
		auto t2 = Clock::now();
		std::cout << "delta time " << std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count() << std::endl;
	}

	Mat out = Mat(1, numRows*numCols, CV_8UC1, h_img1Data_out);

	cout << "out size rows " << out.rows << endl;
	cout << "out size cols " << out.cols << endl;

	cout << "out size height " << out.size().height << endl;
	cout << "out size width " << out.size().width << endl;

	out = out.reshape(1, numRows);

	cout << "out size rows " << out.rows << endl;
	cout << "out size cols " << out.cols << endl;

	cout << "out size height " << out.size().height << endl;
	cout << "out size width " << out.size().width << endl;

	namedWindow("Result", WINDOW_AUTOSIZE);
	imshow("Result", out);
	waitKey(0);
	cin.ignore();

	//hipDeviceSynchronize();

	//Free
	hipFree(d_img1Data_in);
	hipFree(d_img1Data_out);

	

	return 0;
	*/
}