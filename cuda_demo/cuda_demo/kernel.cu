#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <conio.h>
#include <ctime>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>

typedef std::chrono::high_resolution_clock Clock;

//load 2d array
//do something on 2d array
//send back the 2d array

__global__ void cube(float * d_out, float * d_in) {
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f*f*f;
}

__device__ int flatten_idx_gpu(int * matrixIdx, int numRows, int numCols) {
	if (0 <= matrixIdx[0] < numRows && 0 <= matrixIdx[1] < numCols) return matrixIdx[0] * numCols + matrixIdx[1];
	else return 0;
}

__device__ int * unflatten_idx_gpu(int arrayIdx, int numRows, int numCols) {
	if (0 <= arrayIdx < numRows*numCols) {
		int rowIdx = arrayIdx / numCols;
		int idx[] = { rowIdx ,  arrayIdx - rowIdx*numCols };
		return idx;
	}
	return 0;
}

__global__ void rowOperation_block(float * d_matrix1D_out, float * d_matrix1D_in, int numRows, int numCols) {
	//number of blocks should be the number of rows
	//number of threads should be the number of cols
	int idxBlk = blockIdx.x;
	int idxThd = threadIdx.x;
	//give a row per block, in a 2d matrix it would be mat[idxBlk][idxThd], mat[idxBlk][idxThd], mat[idxBlk][idxThd], ...
	int idx2D[] = { idxBlk, idxThd };
	int matIdx = flatten_idx_gpu(idx2D, numRows, numCols);
	d_matrix1D_out[matIdx] = d_matrix1D_in[matIdx] + 1;
}

__global__ void colOperation_block(float * d_matrix1D_out, float * d_matrix1D_in, int numRows, int numCols) {
	int idxBlk = blockIdx.x;
	int idxThd = threadIdx.x;
	//give a column per thread, in a 2d matrix it would be mat[idxThd][idxBlk], mat[idxThd][idxBlk], mat[idxThd][idxBlk], ...
	int idx2D[] = { idxThd, idxBlk };
	int matIdx = flatten_idx_gpu(idx2D, numRows, numCols);
	d_matrix1D_out[matIdx] = d_matrix1D_in[matIdx] + 0.01;
}

__global__ void rowOperation(float * d_matrix1D_out, float * d_matrix1D_in, int numRows, int numCols) {
	int idx = threadIdx.x;
	printf("We are in thread %d\n", idx);
	//give a row per thread, in a 2d matrix it would be mat[idx][0], mat[idx][1], mat[idx][2], ...
	for (int col = 0; col < numCols; col++) {
		int idx2D[] = { idx, col };
		int matIdx = flatten_idx_gpu(idx2D, numRows, numCols);
		d_matrix1D_out[matIdx] = d_matrix1D_in[matIdx] + 1; //in our sample code should increment the rows from 2.14 to 3.14 for instance
	}
}

__global__ void colOperation(float * d_matrix1D_out, float * d_matrix1D_in, int numRows, int numCols) {
	int idx = threadIdx.x;
	//give a column per thread, in a 2d matrix it would be mat[0][idx], mat[1][idx], mat[2][idx], ...
	for (int row = 0; row < numRows; row++) {
		int idx2D[] = { row, idx };
		int matIdx = flatten_idx_gpu(idx2D, numRows, numCols);
		d_matrix1D_out[matIdx] = d_matrix1D_in[matIdx] + 0.01; //in ou sample code should increment the cols from 2.14 to 2.15
	}
}

int flatten_idx(int * matrixIdx, int numRows, int numCols) {
	if (0 <= matrixIdx[0] < numRows && 0 <= matrixIdx[1] < numCols) return matrixIdx[0] * numCols + matrixIdx[1];
	else return 0;
}

int * unflatten_idx(int arrayIdx, int numRows, int numCols) {
	if (0 <= arrayIdx < numRows*numCols) {
		int rowIdx = arrayIdx / numCols;
		int idx[] = { rowIdx ,  arrayIdx - rowIdx*numCols };
		return idx;
	}
	return 0;
}

int main(int argc, char ** argv) {
	const int numRows = 32;
	const int numCols = 16;
	const int ARRAY_SIZE = numRows * numCols;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	//Matrix in 1D indexing
	float h_matrix1D_in[ARRAY_SIZE];

	//Populate
	printf("\n\nPopulate matrix1D(%d,%d) with double for loop, with matrix1D[row*numCols + col] (as used in flatten())...\n", numRows, numCols);
	for (int row = 0; row < numRows; row++) {
		for (int col = 0; col < numCols; col++) {
			h_matrix1D_in[row*numCols + col] = row + (float)col/100;
		}
	}

	//Visualize canonically in 1D
	/*
	printf("\nVisualize matrix1D_in(%d,%d) with one for loop...\n", numRows, numCols);
	for (int i = 0; i < ARRAY_SIZE; i++) {
		if (i % numCols == 0) printf("\n");
		if ((int)h_matrix1D_in[i] < 10) {
			printf(" %.2f ", h_matrix1D_in[i]);
		}
		else {
			printf("%.2f ", h_matrix1D_in[i]);
		}
	}
	printf("\n");
	*/

	float h_matrix1D_out[ARRAY_SIZE];

	// declare GPU memory pointers
	float * d_matrix1D_in;
	float * d_matrix1D_out;

	

	// allocate GPU memory
	hipMalloc((void**)&d_matrix1D_in, ARRAY_BYTES);
	hipMalloc((void**)&d_matrix1D_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_matrix1D_in, h_matrix1D_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	auto t1 = Clock::now();
	auto t2 = Clock::now();
	std::cout << "delta time " << std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count() / 1000000 << std::endl;

	// launch the kernel
	// rowOperation << <1, numRows >> > (d_matrix1D_out, d_matrix1D_in, numRows, numCols);
	// colOperation << <1, numCols >> > (d_matrix1D_out, d_matrix1D_in, numRows, numCols);
	// rowOperation_block << <numRows, numCols >> > (d_matrix1D_out, d_matrix1D_in, numRows, numCols);
	colOperation_block << < numCols, numRows >> > (d_matrix1D_out, d_matrix1D_in, numRows, numCols);
	// cube << <1, ARRAY_SIZE >> > (d_matrix1D_out, d_matrix1D_in);
	
	// copy back the result array to the CPU
	hipMemcpy(h_matrix1D_out, d_matrix1D_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	//Visualize canonically in 1D
	printf("\nVisualize matrix1D_in(%d,%d) with one for loop...\n", numRows, numCols);
	for (int i = 0; i < ARRAY_SIZE; i++) {
		if (i % numCols == 0) printf("\n");
		if ((int)h_matrix1D_out[i] < 10) {
			printf(" %.2f ", h_matrix1D_out[i]);
		}
		else {
			printf("%.2f ", h_matrix1D_out[i]);
		}
	}
	printf("\n");


	hipFree(d_matrix1D_in);
	hipFree(d_matrix1D_out);

	std::cin.ignore();

	return 0;
}