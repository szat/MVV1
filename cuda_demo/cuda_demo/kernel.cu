#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <conio.h>
#include <ctime>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>

typedef std::chrono::high_resolution_clock Clock;

//load 2d array
//do something on 2d array
//send back the 2d array

__global__ void cube(int * d_out, int * d_in) {
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f*f*f;
}

int main(int argc, char ** argv) {
	const int numRows = 64;
	const int numCols = 32;
	const int ARRAY_SIZE = numRows * numCols;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(int);

	// generate the input array on the host
	int h_in[ARRAY_SIZE];
	for (int i = 0; i < numRows; i++) {
		for (int j = 0; j < numCols; j++) {
			h_in[i*numCols + j] = (i) * (j) % 9;
		}
	}
	
	for (int i = 0; i < ARRAY_SIZE; i++) {
		if (i % numCols == 0) printf("\n");
		printf("%i ", h_in[i]);
	}
	printf("\n");

	int h_out[ARRAY_SIZE];

	// declare GPU memory pointers
	int * d_in;
	int * d_out;

	auto t1 = Clock::now();

	// allocate GPU memory
	hipMalloc((void**)&d_in, ARRAY_BYTES);
	hipMalloc((void**)&d_out, ARRAY_BYTES);

	// transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	auto t2 = Clock::now();
	std::cout << "delta time " << std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count() / 1000000 << std::endl;

	// launch the kernel
	cube << <1, ARRAY_SIZE >> > (d_out, d_in);

	// copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// print out the resulting array
	for (int i = 0; i < ARRAY_SIZE; i++) {
		printf("%f", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");
	}

	hipFree(d_in);
	hipFree(d_out);

	std::cin.ignore();

	return 0;
}
