#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
This example demonstrates how to use the Cuda OpenGL bindings to
dynamically modify a vertex buffer using a Cuda kernel.

The steps are:
1. Create an empty vertex buffer object (VBO)
2. Register the VBO with Cuda
3. Map the VBO for writing from Cuda
4. Run Cuda kernel to modify the vertex positions
5. Unmap the VBO
6. Render the results using OpenGL

Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#include <opencv2\imgproc.hpp>
#include <opencv2\highgui.hpp>
#include <string>

using namespace cv;
using namespace std;

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     10 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width = 256;
const unsigned int mesh_height = 256;

// vbo variables
GLuint vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
	unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource);

const char *sSDKsample = "simpleGL (VBO)";

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
	float u = x / (float)width;
	float v = y / (float)height;
	u = u*2.0f - 1.0f;
	v = v*2.0f - 1.0f;

	// calculate simple sine wave pattern
	float freq = 4.0f;
	float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

	// write output vertex
	pos[y*width + x] = make_float4(u, w, v, 1.0f);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
	// map OpenGL buffer object for writing from CUDA
	float4 *dptr;
	checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
		*vbo_resource));
	//printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

	dim3 block(8, 8, 1);
	dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
	simple_vbo_kernel << <grid, block >> > (dptr, mesh_width, mesh_height, g_fAnim);

	// unmap buffer object
	checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{

	string img1_path = "../../data_store/images/david_1.jpg";
	Mat img1 = imread(img1_path, IMREAD_COLOR);


	char *ref_file = NULL;

	pArgc = &argc;
	pArgv = argv;

	printf("%s starting...\n", sSDKsample);

	printf("\n");

	initGL(&argc, argv);
	cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
	// register callbacks
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutCloseFunc(cleanup);
	// create VBO
	createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard); //<== THIS IS IMPORTANT
																	// run the cuda part
	runCuda(&cuda_vbo_resource);
	
	// start rendering mainloop
	glutMainLoop();

	printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
	exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
	glutInit(argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Cuda GL Interop (VBO)");
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);
	glutTimerFunc(REFRESH_DELAY, timerEvent, 0);

	// initialize necessary OpenGL extensions	
	if (!isGLVersionSupported(2, 0))
	{
		fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
		fflush(stderr);
		return false;
	}

	// default initialization
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glDisable(GL_DEPTH_TEST);

	// viewport
	glViewport(0, 0, window_width, window_height);

	// projection
	glMatrixMode(GL_PROJECTION); //of GL_MODELVIEW, for us no diff
	glLoadIdentity();
	gluPerspective(60.0, (GLfloat)window_width / (GLfloat)window_height, 0.1, 10.0);

	SDK_CHECK_ERROR_GL();

	return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res, unsigned int vbo_res_flags)
{
	assert(vbo);

	// create buffer object
	glGenBuffers(1, vbo);
	glBindBuffer(GL_ARRAY_BUFFER, *vbo);

	// initialize buffer object
	unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
	glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	// register this buffer object with CUDA
	checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

	SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{
	// unregister this buffer object with CUDA
	checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

	glBindBuffer(1, *vbo);
	glDeleteBuffers(1, vbo);

	*vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
	// run CUDA kernel to generate vertex positions
	runCuda(&cuda_vbo_resource);

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// set view matrix
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glTranslatef(0.0, 0.0, translate_z);

	// render from the vbo
	glBindBuffer(GL_ARRAY_BUFFER, vbo);
	glVertexPointer(4, GL_FLOAT, 0, 0);

	glEnableClientState(GL_VERTEX_ARRAY);
	glColor3f(1.0, 0.0, 0.0);
	glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);
	glDisableClientState(GL_VERTEX_ARRAY);

	glutSwapBuffers();

	g_fAnim += 0.01f;
}

void timerEvent(int value)
{
	if (glutGetWindow())
	{
		glutPostRedisplay();
		glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
	}
}

void cleanup()
{
	if (vbo)
	{
		deleteVBO(&vbo, cuda_vbo_resource);
	}
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
	switch (key)
	{
	case (27):
		glutDestroyWindow(glutGetWindow());
		return;
	}
}
