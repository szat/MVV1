#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <string>
#include <stdio.h>
#include <iostream>

#include "binary_write.h"
#include "video_preprocessing.h"
#include "interpolate_images.h"
#include "polygon_raster.h"
#include "build_geometry.h"

#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/core/utility.hpp>

#include <opencv2/video.hpp>
//#include <opencv2/optflow.hpp>

//#include <opencv2/features2d/features2d.hpp>
#include <AKAZE.h>
#include <AKAZEConfig.h>
//#include <hip/hip_runtime_api.h>
#include <opencv2/calib3d.hpp> //AKAZE seems not to work without this

#define VERSION "1.0.6"
#define APPLICATION_NAME "MVV"
#define COPYRIGHT_AUTHORS "Adrian Szatmari, Daniel Hogg"
#define COPYRIGHT_YEAR 2017

using namespace std;
using namespace cv;
using namespace libAKAZECU;

const float MIN_H_ERROR = 5.00f;            ///< Maximum error in pixels to accept an inlier
const float DRATIO = 0.80f;

struct GeometricSlice {
	Rect img;
	vector<Vec6f> triangles;
};

struct MatchedGeometry {
	GeometricSlice source_geometry;
	GeometricSlice target_geometry;
};

string pad_frame_number(int frame_number) {
	// zero-padding frame number
	stringstream stream;
	stream << frame_number;
	string padded;
	stream >> padded;
	int str_length = padded.length();
	for (int i = 0; i < 6 - str_length; i++)
		padded = "0" + padded;
	return padded;
}

MatchedGeometry create_matched_geometry(vector<Point2f> imgA_points, vector<Point2f> imgB_points, Size size) {
	// triangulate source interior
	vector<Vec6f> trianglesA = construct_triangles(imgA_points, size);

	// triangulate target interior
	vector<Vec6f> trianglesB = triangulate_target(imgA_points, imgB_points, trianglesA);

	Rect img_bounds = Rect(0, 0, size.width, size.height);

	// This could potentially be replaced by two constructors.
	MatchedGeometry matched_result = MatchedGeometry();
	GeometricSlice source = GeometricSlice();
	GeometricSlice target = GeometricSlice();
	source.triangles = trianglesA;
	target.triangles = trianglesB;
	source.img = img_bounds;
	target.img = img_bounds;
	matched_result.source_geometry = source;
	matched_result.target_geometry = target;
	return matched_result;
}

void ratio_matcher_script(const float ratio, const vector<KeyPoint>& kpts1_in, const vector<KeyPoint>& kpts2_in, const Mat& desc1_in, const Mat& desc2_in, vector<KeyPoint>& kpts1_out, vector<KeyPoint>& kpts2_out) {
	time_t tstart, tend;
	vector<vector<DMatch>> matchesLoweRatio;
	BFMatcher matcher(NORM_HAMMING);
	tstart = time(0);
	matcher.knnMatch(desc1_in, desc2_in, matchesLoweRatio, 2);
	int nbMatches = matchesLoweRatio.size();
	for (int i = 0; i < nbMatches; i++) {
		DMatch first = matchesLoweRatio[i][0];
		float dist1 = matchesLoweRatio[i][0].distance;
		float dist2 = matchesLoweRatio[i][1].distance;
		if (dist1 < ratio * dist2) {
			kpts1_out.push_back(kpts1_in[first.queryIdx]);
			kpts2_out.push_back(kpts2_in[first.trainIdx]);
		}
	}
	tend = time(0);
	cout << "Ratio matching with BF(NORM_HAMMING) and ratio " << ratio << " finished in " << difftime(tend, tstart) << "s and matched " << kpts1_out.size() << " features." << endl;
}

void ransac_script(const float ball_radius, const float inlier_thresh, const vector<KeyPoint>& kpts1_in, const vector<KeyPoint>& kpts2_in, Mat& homography_out, vector<KeyPoint>& kpts1_out, vector<KeyPoint>& kpts2_out) {
	cout << "RANSAC to estimate global homography with max deviating distance being " << ball_radius << "." << endl;

	vector<Point2f> keysImage1;
	vector<Point2f> keysImage2;
	vector<DMatch> good_matches;

	int nbMatches = kpts1_in.size();
	for (int i = 0; i < nbMatches; i++) {
		keysImage1.push_back(kpts1_in.at(i).pt);
		keysImage2.push_back(kpts2_in.at(i).pt);
	}

	Mat H = findHomography(keysImage1, keysImage2, CV_RANSAC, ball_radius);
	homography_out = H;

	cout << "RANSAC found the homography." << endl;

	nbMatches = kpts1_in.size();
	for (int i = 0; i < nbMatches; i++) {
		Mat col = Mat::ones(3, 1, CV_64F);// , CV_32F);
		col.at<double>(0) = kpts1_in[i].pt.x;
		col.at<double>(1) = kpts1_in[i].pt.y;

		col = H * col;
		col /= col.at<double>(2); //because you are in projective space
		double dist = sqrt(pow(col.at<double>(0) - kpts2_in[i].pt.x, 2) + pow(col.at<double>(1) - kpts2_in[i].pt.y, 2));

		if (dist < inlier_thresh) {
			int new_i = static_cast<int>(kpts1_out.size());
			kpts1_out.push_back(kpts1_in[i]);
			kpts2_out.push_back(kpts2_in[i]);
		}
	}

	cout << "Homography filtering with inlier threshhold of " << inlier_thresh << " has matched " << kpts1_out.size() << " features." << endl;
}

vector<vector<KeyPoint>> match_points_mat(Mat img1, Mat img2)
{
	const float akaze_thr = 3e-4;    // AKAZE detection threshold set to locate about 1000 keypoints
	const float ratio = 0.8f;   // Nearest neighbor matching ratio
	const float inlier_thr = 20.0f; // Distance threshold to identify inliers
	const float ball_radius = 5;

	vector<KeyPoint> kpts1_step1;
	vector<KeyPoint> kpts2_step1;
	Mat desc1_step1;
	Mat desc2_step1;

	//akaze_script(akaze_thresh, img1, kpts1_step1, desc1_step1);
	//akaze_script(akaze_thresh, img2, kpts2_step1, desc2_step1);
	
	

	vector<KeyPoint> kpts1_step2;
	vector<KeyPoint> kpts2_step2;
	ratio_matcher_script(ratio, kpts1_step1, kpts2_step1, desc1_step1, desc2_step1, kpts1_step2, kpts2_step2);

	Mat homography;
	vector<KeyPoint> kpts1_step3;
	vector<KeyPoint> kpts2_step3;
	ransac_script(ball_radius, inlier_thr, kpts1_step2, kpts2_step2, homography, kpts1_step3, kpts2_step3);

	vector<vector<KeyPoint>> pointMatches = { kpts1_step3, kpts2_step3 };
	return pointMatches;
}

MatchedGeometry read_matched_points_from_file(Mat &img1, Mat &img2, Size video_size) {
	cout << "Initializing matched geometry routine" << endl;

	Mat imgA;
	Mat imgB;
	cvtColor(img1, imgA, CV_BGR2GRAY);
	cvtColor(img2, imgB, CV_BGR2GRAY);

	vector<vector<KeyPoint>> point_matches = match_points_mat(imgA, imgB);

	vector<KeyPoint> imgA_keypoints = point_matches[0];
	vector<KeyPoint> imgB_keypoints = point_matches[1];
	vector<Point2f> imgA_points = convert_key_points(imgA_keypoints);
	vector<Point2f> imgB_points = convert_key_points(imgB_keypoints);

	MatchedGeometry geometry = create_matched_geometry(imgA_points, imgB_points, video_size);
	return geometry;
}

void save_frame_master(Mat &img1, Mat &img2, Size video_size, string affine, string rasterA, string rasterB) {
	MatchedGeometry geometry = read_matched_points_from_file(img1, img2, video_size);

	vector<Vec6f> trianglesA = geometry.source_geometry.triangles;
	vector<Vec6f> trianglesB = geometry.target_geometry.triangles;

	Rect imgA_bounds = geometry.source_geometry.img;
	Rect imgB_bounds = geometry.target_geometry.img;

	vector<vector<Point>> rastered_trianglesA = raster_triangulation(trianglesA, imgA_bounds);
	vector<vector<Point>> rastered_trianglesB = raster_triangulation(trianglesB, imgB_bounds);

	int widthA = imgA_bounds.width;
	int heightA = imgA_bounds.height;
	int widthB = imgB_bounds.width;
	int heightB = imgB_bounds.height;

	// save affine params as .csv
	// save image raster as grayscale .png from 0-65536 (2 images)
	short** gridA = grid_from_raster(widthA, heightA, rastered_trianglesA);
	short** gridB = grid_from_raster(widthB, heightB, rastered_trianglesB);
	save_raster(rasterA, gridA, widthA, heightA);
	save_raster(rasterB, gridB, widthB, heightB);

	vector<Mat> affine_forward = get_affine_transforms_forward(trianglesA, trianglesB);
	vector<Mat> affine_reverse = get_affine_transforms_reverse(trianglesB, trianglesA, affine_forward);

	float* affine_params = convert_vector_params(affine_forward, affine_reverse);
	write_float_array(affine, affine_params, trianglesA.size() * 12);
}

int video_loop(VideoCapture & cap_1, VideoCapture & cap_2, int start_1, int start_2) {

	// do the point matching at max resolution, then rescale
	// doens't seem like we do any rescaling

	int starter_offset = 10;

	start_1 = start_1 + starter_offset;
	start_2 = start_2 + starter_offset;

	int num_frames_1 = cap_1.get(CV_CAP_PROP_FRAME_COUNT);
	int num_frames_2 = cap_2.get(CV_CAP_PROP_FRAME_COUNT);

	int width_1 = cap_1.get(CV_CAP_PROP_FRAME_WIDTH);
	int height_1 = cap_1.get(CV_CAP_PROP_FRAME_HEIGHT);

	int width_2 = cap_2.get(CV_CAP_PROP_FRAME_WIDTH);
	int height_2 = cap_2.get(CV_CAP_PROP_FRAME_HEIGHT);

	if (width_1 != width_2 || height_1 != height_2) {
		cout << "ERROR" << endl;
		return 0;
	}

	Size video_size = Size(width_1, height_1);

	cap_1.set(CV_CAP_PROP_POS_FRAMES, start_1);
	cap_2.set(CV_CAP_PROP_POS_FRAMES, start_2);

	Mat next_1;
	Mat next_2;

	int frames_remaining_1 = num_frames_1 - start_1 - 1;
	int frames_remaining_2 = num_frames_2 - start_2 - 1;
	int frames_remaining = min(frames_remaining_1, frames_remaining_2);

	// Determining how many 'jumps' are required.
	// TODO: Replace these variables with more descriptive and intuitive names.

	int renderable_frames = frames_remaining - frames_remaining % 20;
	int jump_size = 20;
	int num_jumps = renderable_frames / 20;
	int cutoff_frame = jump_size * num_jumps;

	for (int i = 0; i <= cutoff_frame; i += jump_size) {
		string padded_number = pad_frame_number(i);
		cout << "Processing frame " << i << " of " << cutoff_frame << endl;

		string affine_dir = "../../data_store/affine/";
		string filename_affine = "affine_" + padded_number + ".bin";

		string raster_dir = "../../data_store/raster/";
		string filename_raster_1 = "raster_1_" + padded_number + ".bin";
		string filename_raster_2 = "raster_2_" + padded_number + ".bin";

		string image_dir = "../../data_store/binary/";
		string filename_img_1 = "img1_" + padded_number + ".bin";
		string filename_img_2 = "img2_" + padded_number + ".bin";

		string affine = affine_dir + filename_affine;
		string raster1 = raster_dir + filename_raster_1;
		string raster2 = raster_dir + filename_raster_2;
		string img1;
		string img2;

		cap_1.read(next_1);
		cap_2.read(next_2);

		save_frame_master(next_1, next_2, video_size, affine, raster1, raster2);

		cout << "Saving image for frame " << i << endl;
		padded_number = pad_frame_number(i);
		filename_img_1 = "img1_" + padded_number + ".bin";
		filename_img_2 = "img2_" + padded_number + ".bin";
		img1 = image_dir + filename_img_1;
		img2 = image_dir + filename_img_2;
		save_img_binary(next_1, next_2, video_size, img1, img2);

		for (int j = 1; j < 20; j++) {
			cout << "Saving image for frame " << (i + j) << endl;
			cap_1.read(next_1);
			cap_2.read(next_2);
			padded_number = pad_frame_number(i + j);
			filename_img_1 = "img1_" + padded_number + ".bin";
			filename_img_2 = "img2_" + padded_number + ".bin";
			img1 = image_dir + filename_img_1;
			img2 = image_dir + filename_img_2;
			save_img_binary(next_1, next_2, video_size, img1, img2);
		}
	}
	return -1;
}

int main() {
	// Initializing application
	cout << APPLICATION_NAME << " version " << VERSION << endl;
	cout << COPYRIGHT_AUTHORS << " " << COPYRIGHT_YEAR << ". " << "MIT License." << endl;

	// Likely, although this is subject to debate, there should be a command-line interface
	// that prompts the user to enter the video file names and other input parameters, and does error
	// checking to make sure everything is valid.

	// KEY POINT #1: The program should fail gracefully if the input data is improper!
	// KEY POINT #2: The program should exit with the first error code it encounters and print to the console. Although, for this
	// part of the program, maybe it should just go in a loop until you get it right.

	// ERROR CODE 001: File name must contain a file extension of the form .mp4, .avi, etc. (all video types we support should be clearly listed)
	// ERROR CODE 002: Video file not found. Please verify the file specified exists in the data_store/video folder (or whatever folder we use)
	// ERROR CODE 003: The 'start_offset' parameter, which specifies how many frames (if any) are to be skipped before processing the video, cannot be negative.
	// ERROR CODE 004: The 'start_offset' parameter cannot be greater than the number of frames in either of the two input videos.
	// ERROR CODE 005: The 'delay' parameter must be a parsable positive floating point number (for example, 6.2657 is a valid input). 
	// ERROR CODE 006: The 'delay' parameter specified was too large and did not yield any usable frames (no overlap).
	// ERROR CODE 007: The framerate must be a positive integer.

	string video_path_1 = "C:\\Users\\Danny\\Documents\\GitHub\\mvv\\data_store\\video\\judo_left.MP4";
	string video_path_2 = "C:\\Users\\Danny\\Documents\\GitHub\\mvv\\data_store\\video\\judo_right.MP4";
	int start_offset = 500;
	float delay = 6.2657f;
	int framerate = 95;

	VideoCapture cap_1(video_path_1);
	if (!cap_1.isOpened()) {
		cout << "Video 1 failed to load." << endl;
		return -1;
	}

	VideoCapture cap_2(video_path_2);
	if (!cap_2.isOpened()) {
		cout << "Video 2 failed to load." << endl;
		return -1;
	}

	pair<int, int> initial_offset = audio_sync(start_offset, delay, framerate);
	video_loop(cap_1, cap_2, initial_offset.first, initial_offset.second);

	/*

	int num_frames_1 = cap_1.get(CAP_PROP_FRAME_COUNT);
	int num_frames_2 = cap_2.get(CAP_PROP_FRAME_COUNT);

	int start = 500;
	int offset = 595;
	cap_1.set(CV_CAP_PROP_POS_FRAMES, start + offset);
	cap_2.set(CV_CAP_PROP_POS_FRAMES, start);

	Mat img1;
	cap_1.read(img1);
	img1 = img1.clone();
	Mat img2;
	cap_2.read(img2);
	img2 = img2.clone();

	imwrite("..\\data_store\\images\\judo_left.png", img1);
	imwrite("..\\data_store\\images\\judo_right.png", img2);

	//So this works well
	AKAZEOptions options;

	// Convert the image to float to extract features
	Mat img1_gray;
	cvtColor(img1, img1_gray, CV_BGR2GRAY);
	Mat img2_gray;
	cvtColor(img2, img2_gray, CV_BGR2GRAY);
	Mat img1_32;
	img1_gray.convertTo(img1_32, CV_32F, 1.0 / 255.0, 0);
	Mat img2_32;
	img2_gray.convertTo(img2_32, CV_32F, 1.0 / 255.0, 0);

	// Don't forget to specify image dimensions in AKAZE's options
	options.img_width = img1.cols;
	options.img_height = img1.rows;

	// Extract features
	libAKAZECU::AKAZE evolution(options);
	vector<KeyPoint> kpts1;
	vector<KeyPoint> kpts2;
	vector<vector<cv::DMatch> > dmatches;
	Mat desc1;
	Mat desc2;

	evolution.Create_Nonlinear_Scale_Space(img1_32);
	evolution.Feature_Detection(kpts1);
	evolution.Compute_Descriptors(kpts1, desc1);

	evolution.Create_Nonlinear_Scale_Space(img2_32);
	evolution.Feature_Detection(kpts2);
	evolution.Compute_Descriptors(kpts2, desc2);

	Matcher cuda_matcher;

	cuda_matcher.bfmatch(desc1, desc2, dmatches);
	cuda_matcher.bfmatch(desc2, desc1, dmatches);

	vector<cv::Point2f> matches, inliers;

	matches2points_nndr(kpts2, kpts1, dmatches, matches, DRATIO);
	compute_inliers_ransac(matches, inliers, MIN_H_ERROR, false);

	Mat img_com = cv::Mat(cv::Size(img1.cols * 2, img1.rows), CV_8UC3);
	draw_keypoints(img1, kpts1);
	draw_keypoints(img2, kpts2);
	draw_inliers(img1, img2, img_com, inliers);
	cv::namedWindow("Inliers", cv::WINDOW_NORMAL);
	cv::imshow("Inliers", img_com);
	cv::waitKey(0);

	cout << "TESTING 001" << endl;

	*/

	cin.get();

	return 0;
}