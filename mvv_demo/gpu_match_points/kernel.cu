#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void akaze_script(float akaze_thresh, const Mat& img_in, vector<KeyPoint>& kpts_out, Mat& desc_out) {
	Ptr<AKAZE> akaze = AKAZE::create();
	akaze->setThreshold(akaze_thresh);
	time_t tstart, tend;
	tstart = time(0);
	akaze->detectAndCompute(img_in, noArray(), kpts_out, desc_out);
	tend = time(0);
	cout << "akaze_wrapper(thr=" << akaze_thresh << ",[h=" << img_in.size().height << ",w=" << img_in.size().width << "]) finished in " << difftime(tend, tstart) << "s and found " << kpts_out.size() << " features." << endl;
}

