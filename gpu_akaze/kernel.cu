
#include "hip/hip_runtime.h"
#include ""

#include <string>
#include <stdio.h>
#include <iostream>

#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/core/utility.hpp>

#include <opencv2/video.hpp>
//#include <opencv2/optflow.hpp>

//#include <opencv2/features2d/features2d.hpp>
#include <AKAZE.h>
#include <AKAZEConfig.h>
//#include <hip/hip_runtime_api.h>
#include <opencv2/calib3d.hpp> //AKAZE seems not to work without this

using namespace std;
using namespace cv;
using namespace libAKAZECU;

const float MIN_H_ERROR = 5.00f;            ///< Maximum error in pixels to accept an inlier
const float DRATIO = 0.80f;

int main() {
	VideoCapture cap_1("..\\data_store\\judo\\judo_left.MP4");
	if (!cap_1.isOpened()) {
		cout << "Video 1 failed to load." << endl;
		return -1;
	}

	VideoCapture cap_2("..\\data_store\\judo\\judo_right.MP4");
	if (!cap_2.isOpened()) {
		cout << "Video 1 failed to load." << endl;
		return -1;
	}

	int num_frames_1 = cap_1.get(CAP_PROP_FRAME_COUNT);
	int num_frames_2 = cap_2.get(CAP_PROP_FRAME_COUNT);

	int start = 500;
	int offset = 595;
	cap_1.set(CV_CAP_PROP_POS_FRAMES, start+offset);
	cap_2.set(CV_CAP_PROP_POS_FRAMES, start);

	Mat img1;
	cap_1.read(img1);
	img1 = img1.clone();
	Mat img2;
	cap_2.read(img2);
	img2 = img2.clone();

	imwrite("..\\data_store\\images\\judo_left.png", img1);
	imwrite("..\\data_store\\images\\judo_right.png", img2);

	//So this works well
	AKAZEOptions options;

	// Convert the image to float to extract features
	Mat img1_gray;
	cvtColor(img1, img1_gray, CV_BGR2GRAY);
	Mat img2_gray;
	cvtColor(img2, img2_gray, CV_BGR2GRAY);
	Mat img1_32;
	img1_gray.convertTo(img1_32, CV_32F, 1.0 / 255.0, 0);
	Mat img2_32;
	img2_gray.convertTo(img2_32, CV_32F, 1.0 / 255.0, 0);

	// Don't forget to specify image dimensions in AKAZE's options
	options.img_width = img1.cols;
	options.img_height = img1.rows;

	// Extract features
	libAKAZECU::AKAZE evolution(options);
	vector<KeyPoint> kpts1;
	vector<KeyPoint> kpts2;
	vector<vector<cv::DMatch> > dmatches;
	Mat desc1;
	Mat desc2;

	evolution.Create_Nonlinear_Scale_Space(img1_32);
	evolution.Feature_Detection(kpts1);
	evolution.Compute_Descriptors(kpts1, desc1);

	evolution.Create_Nonlinear_Scale_Space(img2_32);
	evolution.Feature_Detection(kpts2);
	evolution.Compute_Descriptors(kpts2, desc2);

	Matcher cuda_matcher;

	cuda_matcher.bfmatch(desc1, desc2, dmatches);
	cuda_matcher.bfmatch(desc2, desc1, dmatches);

	vector<cv::Point2f> matches, inliers;

	matches2points_nndr(kpts2, kpts1, dmatches, matches, DRATIO);
	compute_inliers_ransac(matches, inliers, MIN_H_ERROR, false);

	Mat img_com = cv::Mat(cv::Size(img1.cols * 2, img1.rows), CV_8UC3);
	draw_keypoints(img1, kpts1);
	draw_keypoints(img2, kpts2);
	draw_inliers(img1, img2, img_com, inliers);
	cv::namedWindow("Inliers", cv::WINDOW_NORMAL);
	cv::imshow("Inliers", img_com);
	cv::waitKey(0);

	return 0;
}